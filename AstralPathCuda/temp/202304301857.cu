
#include <hip/hip_runtime.h>
#include<stdio.h>
using namespace std;
#include<hiprand/hiprand.h>
//现在开始全局变量书写
//现在开始全局内存书写
int main(int argc, char* argv[]) {
        hiprandGenerator_t generator ;
        hiprandCreateGenerator(&generator, HIPRAND_RNG_QUASI_SOBOL32);
        hiprandSetPseudoRandomGeneratorSeed(generator, 1234);
        int N = 100000;
        double *g_x;
        hipMalloc((void **)&g_x, sizeof(double) * N);
        hiprandGenerateUniformDouble(generator, g_x, N);
        double *x = (double*) calloc(N, sizeof(double));
        hipMemcpy(x, g_x, sizeof(double) * N, hipMemcpyDeviceToHost);

        for (int y = 0; y < N ;y ++) {
            printf("%g\n",x[y]);
        }
        hipFree(g_x);
        free(x);
}
