#include<stdio.h>
#include <hip/hip_runtime.h>
using namespace std;
//现在开始全局变量书写
     const int N = 100;//global
//现在开始全局内存书写
__device__ void add(int *a,int *b,int *z) {
        z[threadIdx.x] = a[threadIdx.x]+ b[threadIdx.x];
}
__global__ void hello(int *h,int *b,int *z) {
        int n = threadIdx.x;
        add(h,b,z);
        printf("(%d,%d) = %d \n",threadIdx.x,threadIdx.y,z[n]);
        b[n] = h[n];

}
__global__ void hello2(int *h,int *b,int *z) {
        int n = threadIdx.x;
        add(h,b,z);
        printf("(%d,%d) = %d \n",threadIdx.x,threadIdx.y,z[n]);
        b[n] = h[n];

}
int main(int argc, char* argv[]) {
        int M = sizeof(int)*N;
        int *h_x = (int *) malloc(M);
        for (int x = 0;x < N;x ++) {
            h_x[x] = 1;
        }
        int *d_x1;
        int *d_y1;
        int *d_y2;

        hipMalloc((void **)&d_x1,M);
        hipMemcpy(d_x1,h_x,M,hipMemcpyHostToDevice);
        hipMalloc((void **)&d_y1,M);
        hipMemcpy(d_y1,h_x,M,hipMemcpyHostToDevice);
        hipMalloc((void **)&d_y2,M);

        const int block_size = 128;
        const int grid_size =  (N + block_size - 1) / block_size;

        hello<<<grid_size,block_size>>>(d_x1,d_y1,d_y2);

        hipDeviceSynchronize();

        hipMemcpy(h_x,d_y1,M,hipMemcpyDeviceToHost);


        free(h_x);
        hipFree(d_x1);

        hipFree(d_y1);

}
