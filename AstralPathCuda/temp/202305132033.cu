
#include <hip/hip_runtime.h>
#include<stdio.h>
using namespace std;
//现在开始全局变量书写
     const int N = 1000;//global
//现在开始全局内存书写
__global__ void test(float *h,float *b,int x) {
        printf("Stream: %d - (%f,%f)\n",x,threadIdx.x,threadIdx.y);
        b[threadIdx.x] = sqrt(h[threadIdx.x]);
    }
int main(int argc, char* argv[]) {
        hipStream_t stream_1 ;
        hipStream_t stream_2 ;

        hipStreamCreate(&stream_1);
        hipStreamCreate(&stream_2);

        int M = sizeof(float)*N;
        float *h_x = (float*) malloc(M);
        for (int x = 0;x < N;x ++) {
            h_x[x] = (float) ((x + 1) * 999.0123);
        }
        float *d_x1;
        float *d_y1;
        hipMalloc((void **)&d_x1,M);
        hipMemcpy(d_x1,h_x,M,hipMemcpyHostToDevice);
        hipMalloc((void **)&d_y1,M);

        const int block_size = 128;
        const int grid_size =  (N + block_size - 1) / block_size;

        test<<<grid_size,block_size,0,stream_1>>>(d_x1,d_y1,0);//tags:<<<grid_size,block_size,0,stream_1>>>
        test<<<grid_size,block_size,0,stream_2>>>(d_x1,d_y1,1);//tags:<<<grid_size,block_size,0,stream_2>>>

        hipStreamSynchronize(stream_1);
        hipStreamSynchronize(stream_2);

        hipStreamDestroy(stream_1);
        hipStreamDestroy(stream_2);
}
