
#include <hip/hip_runtime.h>
#include<stdio.h>
using namespace std;
//现在开始全局变量书写
     const int N = 10000;//global
//现在开始全局内存书写
__device__  double y[N] =  {};//length:N
__device__  double y2[N] =  {};//length:N
__device__  double xy[N] =  {};//length:N
__device__  double x2[N] =  {};//length:N
__device__  double x[N] =  {};//length:N
__device__  double temp[10] =  {};//length:10
__global__ void cor(int limit) {
        const int tid = threadIdx.x;
        const int bid = blockIdx.x;
        const int n = tid;
extern __shared__ double __shared__x[];
extern __shared__ double __shared__x2[];
extern __shared__ double __shared__y[];
extern __shared__ double __shared__y2[];
extern __shared__ double __shared__xy[];
        if (n < limit) {
            __shared__x[tid] = __ldg(&x[n]);
            __shared__y[tid] = __ldg(&y[n]);
            __shared__x2[tid] = __shared__x[tid] * __shared__x[tid];
            __shared__y2[tid] = __shared__y[tid] * __shared__y[tid];
            __shared__xy[tid] = __shared__x[tid] * __shared__y[tid];
        }
        __syncthreads();

        for (int offset = blockDim.x >> 1; offset >= 32; offset >>= 1)
        {
            if (tid < offset)
            {
                __shared__x[tid] += __shared__x[tid + offset];
                __shared__x2[tid] += __shared__x2[tid + offset];
                __shared__y[tid] += __shared__y[tid + offset];
                __shared__y2[tid] += __shared__y2[tid + offset];
                __shared__xy[tid] += __shared__xy[tid + offset];
            }
            __syncthreads();
        }
        double dx = __shared__x[tid];
        double dx2 = __shared__x2[tid];
        double dy = __shared__y[tid];
        double dy2 = __shared__y2[tid];
        double dxy = __shared__xy[tid];
        for (int offset = 16; offset > 0; offset >>= 1)
        {
            dx += __shfl_down_sync(0xffffffff, dx, offset);
            dx2 += __shfl_down_sync(0xffffffff, dx2, offset);
            dy += __shfl_down_sync(0xffffffff, dy, offset);
            dy2 += __shfl_down_sync(0xffffffff, dy2, offset);
            dxy += __shfl_down_sync(0xffffffff, dxy, offset);
        }

        if (tid == 0)
        {
            atomicAdd(&temp[0], dx);
            atomicAdd(&temp[1], dx2);
            atomicAdd(&temp[2], dy);
            atomicAdd(&temp[3], dy2);
            atomicAdd(&temp[4], dxy);
        }
        if (n == 0) {
            dx = temp[0];
            dx2 = temp[1];
            dy = temp[2];
            dy2 = temp[3];
            dxy = temp[4];

            double fenzi = dxy - ((dx * dy) / limit);
            double fenmu = sqrt((dx2 - (dx / limit)) * (dy2 - (dy / limit)));
            temp[5] = fenzi / fenmu;
        }
    }
int main(int argc, char* argv[]) {
        double hx[N] = {};
        double hy[N] = {};
        for(int cx = 0; cx < N ; ++cx) {
            hx[cx] = 1.000 * cx;
            hy[cx] = 2.000 * cx;
        }
        hipMemcpyToSymbol(HIP_SYMBOL(x),hx,sizeof(double)*N);
        hipMemcpyToSymbol(HIP_SYMBOL(y),hy,sizeof(double)*N);

        const int block_size = 128;
        const int grid_size =  (N + block_size - 1) / block_size;
        cor<<<block_size,grid_size,128>>>(N);//tags:<<<block_size,grid_size,128>>>


        int device_id = 0;
        hipMemcpyFromSymbol(hx,HIP_SYMBOL(temp),sizeof(double)*2);
        printf("c=%d\n",hx[0]);
}
