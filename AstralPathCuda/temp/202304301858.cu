#include<stdio.h>
using namespace std;
#include<hiprand.h>
//现在开始全局变量书写
//现在开始全局内存书写
int main(int argc, char* argv[]) {
        hiprandGenerator_t generator ;
        hiprandCreateGenerator(&generator, HIPRAND_RNG_QUASI_SOBOL32);
        hiprandSetPseudoRandomGeneratorSeed(generator, 1234);
        int N = 100000;
        int *g_x;
        hipMalloc((void **)&g_x, sizeof(int) * N);
        curandGenerateNormalInt(generator, g_x, N, 200,10);
        int *x = (int*) calloc(N, sizeof(int));
        hipMemcpy(x, g_x, sizeof(int) * N, hipMemcpyDeviceToHost);

        for (int y = 0; y < N ;y ++) {
            printf("%g\n",x[y]);
        }
        hipFree(g_x);
        free(x);
}
