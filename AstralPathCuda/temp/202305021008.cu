
#include <hip/hip_runtime.h>
#include<stdio.h>
using namespace std;
//现在开始全局变量书写
     const int N = 1000;//global
//现在开始全局内存书写
__device__  int d[2] =  {};//length:2
__device__  int d_x[N] =  {};//length:N
__global__ void reduce() {
        const int tid = threadIdx.x;
        const int bid = blockIdx.x;
        const int n = tid;
extern __shared__ int __shared__s_y[];
        if (n < N) {
            __shared__s_y[tid] = __ldg(&d_x[n]);
        }
        __syncthreads();

        for (int offset = blockDim.x >> 1; offset >= 32; offset >>= 1)
        {
            if (tid < offset)
            {
                __shared__s_y[tid] += __shared__s_y[tid + offset];
            }
            __syncthreads();
        }

        int y = __shared__s_y[tid];
        for (int offset = 16; offset > 0; offset >>= 1)
        {
            y += __shfl_down_sync(0xffffffff, y, offset);
        }

        if (tid == 0)
        {
            atomicAdd(&d[0], y);
        }
    }
int main(int argc, char* argv[]) {
        int ha[N] = {};
        for(int x = 0; x < N ; ++x) {
            ha[x] = 1;
        }
        ha[0] = atoi(argv[1]);

        hipMemcpyToSymbol(HIP_SYMBOL(d_x),ha,sizeof(int)*N);

        const int block_size = 128;
        const int grid_size =  (N + block_size - 1) / block_size;

        reduce<<<block_size,grid_size,128>>>();//tags:<<<block_size,grid_size,128>>>


        int device_id = 0;
        hipMemcpyFromSymbol(ha,HIP_SYMBOL(d),sizeof(int)*2);
        printf("c=%d\n",ha[0]);
}
