
#include <hip/hip_runtime.h>
#include<stdio.h>
using namespace std;
//现在开始全局变量书写
     const int N = 100;//global
//现在开始全局内存书写
__global__ void hello(int *h,int *b) {
        int n = blockDim.x * blockIdx.x + threadIdx.x;
        printf("(%d,%d,%d)\n",blockIdx.x,blockIdx.y,blockIdx.z);
        b[n] = h[n];
}
int main(int argc, char* argv[]) {
        int M = sizeof(int)*N;
        int *h_x = (int*) malloc(M);
        int *h;
        
        for (int x = 0;x < N;x ++) {
            h_x[x] = 1;
        }
        int *d_x1;
        int *d_x2;
        int *d_x3;
        int *d_y1;
        int *d_y2;
        int *d_y3;
        hipMalloc((void **)&d_x1,M);
        hipMalloc((void **)&d_x2,M);
        hipMalloc((void **)&d_x3,M);
        hipMemcpy(d_x1,h_x,M,hipMemcpyHostToDevice);
        hipMemcpy(d_x2,h_x,M,hipMemcpyHostToDevice);
        hipMemcpy(d_x3,h_x,M,hipMemcpyHostToDevice);
        hipMalloc((void **)&d_y1,M);
        hipMalloc((void **)&d_y2,M);
        hipMalloc((void **)&d_y3,M);

        const int block_size = 128;
        const int grid_size =  (N + block_size - 1) / block_size;
        hipStream_t stream_t1 ;
        hipStream_t stream_t2 ;
        hipStream_t stream_t3 ;
        hipStreamCreate(&stream_t1);
        hipStreamCreate(&stream_t2);
        hipStreamCreate(&stream_t3);
        hello<<<grid_size,block_size,0,stream_t1>>>(d_x1,d_y1);//[tags:<<<grid_size,block_size,0,stream_t1>>>]
        hello<<<grid_size,block_size,0,stream_t2>>>(d_x2,d_y2);//[tags:<<<grid_size,block_size,0,stream_t2>>>]
        hello<<<grid_size,block_size,0,stream_t3>>>(d_x3,d_y3);//[tags:<<<grid_size,block_size,0,stream_t3>>>]
        hipDeviceSynchronize();
        hipStreamSynchronize(stream_t1);
        hipStreamSynchronize(stream_t2);
        hipStreamSynchronize(stream_t3);
        hipMemcpy(h_x,d_y1,M,hipMemcpyDeviceToHost);


        free(h_x);
        hipFree(d_x1);
        hipFree(d_x2);
        hipFree(d_x3);
        hipFree(d_y1);
        hipFree(d_y2);
        hipFree(d_y3);
        hipStreamDestroy(stream_t1);
        hipStreamDestroy(stream_t2);
        hipStreamDestroy(stream_t3);
}
