
#include <hip/hip_runtime.h>
#include<stdio.h>
using namespace std;
//现在开始全局变量书写
     const int N = 5;//global
//现在开始全局内存书写
__device__  double d_z[10] =  {};//length:10
__global__ void sum(double *d_x,double *d_y) {
        const int tid = threadIdx.x;
        const int bid = blockIdx.x;
__shared__ double __shared__s_x[128];
__shared__ double __shared__s_x2[128];
__shared__ double __shared__s_y[128];
__shared__ double __shared__s_xy[128];
        const int n = bid * blockDim.x + tid;
        double x = 0.0;
        double y = 0.0;
        double x2 = 0.0;
        double xy = 0.0;
        __shared__s_x[tid] = (n < N) ? d_x[n] : 0.0;
        __shared__s_x2[tid] = (n < N) ? (__shared__s_x[tid] * __shared__s_x[tid]) : 0.0;
        __shared__s_y[tid] = (n < N) ? d_y[n] : 0.0;

        __shared__s_xy[tid] = (n < N) ? __shared__s_x[tid] * __shared__s_y[tid] : 0.0;
        __syncthreads();
        for (int offset = blockDim.x >> 1; offset >= 32; offset >>= 1)
        {
            if (tid < offset)
            {
                __shared__s_x[tid] += __shared__s_x[tid + offset];
                __shared__s_y[tid] += __shared__s_y[tid + offset];
                __shared__s_x2[tid] += __shared__s_x2[tid + offset];
                __shared__s_xy[tid] += __shared__s_xy[tid + offset];
            }
            __syncthreads();
        }

        y = __shared__s_y[tid];
        x = __shared__s_x[tid];
        x2 = __shared__s_x2[tid];
        xy = __shared__s_xy[tid];
        for (int offset = 16; offset > 0; offset >>= 1)
        {
            y += __shfl_down_sync(0xffffffff,y, offset);
            x += __shfl_down_sync(0xffffffff,x, offset);
            x2 += __shfl_down_sync(0xffffffff,x2, offset);
            xy += __shfl_down_sync(0xffffffff,xy, offset);
        }

        if (tid == 0)
        {
            atomicAdd(&d_z[0], x);
            atomicAdd(&d_z[1], y);
            atomicAdd(&d_z[2], x2);
            atomicAdd(&d_z[3], xy);

        }

        if(n == 0) {
            double fenzi = d_z[3] - (d_z[0] * d_z[1] / N);
            double fenmu = d_z[2] - (d_z[0] * d_z[0] / N);
            d_z[4] = fenzi / fenmu;
        }
    }
int main(int argc, char* argv[]) {
        double *a = (double *) malloc(sizeof(double)*N);
        double *a2 = (double *) malloc(sizeof(double)*N);
        double *b = (double *) malloc(sizeof(double)*10);
        for (int x = 0 ;x < N;x ++) {
            a[x] = 11 + 1.0 * x;
            a2[x] = 11 + 3.0124 * x;
        }
        double *d_x;
        double *d_y;
        hipMalloc((void **)&d_x,sizeof(double)*N);
        hipMalloc((void **)&d_y,sizeof(double)*N);
        hipMemcpy(d_x,a,sizeof(double)*N,hipMemcpyHostToDevice);
        hipMemcpy(d_y,a2,sizeof(double)*N,hipMemcpyHostToDevice);
        const int block_size = 128;
        const int grid_size =  (N + block_size - 1) / block_size;
        sum<<<grid_size,block_size>>>(d_x,d_y);
        hipDeviceSynchronize();
        hipMemcpyFromSymbol(b,HIP_SYMBOL(d_z),sizeof(double)*10);
        printf("%f \n",b[0]);
        printf("%f \n",b[1]);
        printf("%f \n",b[2]);
        printf("%f \n",b[3]);
        printf("%f \n",b[4]);
}
