#include<stdio.h>
#include <hip/hip_runtime.h>
using namespace std;
//现在开始全局变量书写
     const int N = 100000;//global
//现在开始全局内存书写
__device__  double d_z[10] =  {};//length:10
__global__ void sum(double *d_x,double *d_y) {
        const int tid = threadIdx.x;
        const int bid = blockIdx.x;
__shared__ double __shared__s_x[128];
__shared__ double __shared__s_x2[128];
__shared__ double __shared__s_y[128];
__shared__ double __shared__s_xy[128];
        const int n = bid * blockDim.x + tid;
        double x = 0.0;
        double y = 0.0;
        double x2 = 0.0;
        double xy = 0.0;
        __shared__s_x[tid] = (n < N) ? d_x[n] : 0.0;
        __shared__s_x2[tid] = (n < N) ? (__shared__s_x[tid] * __shared__s_x[tid]) : 0.0;
        __shared__s_y[tid] = (n < N) ? d_y[n] : 0.0;

        __shared__s_xy[tid] = (n < N) ? __shared__s_x[tid] * __shared__s_y[tid] : 0.0;
        __syncthreads();
        for (int offset = blockDim.x >> 1; offset >= 32; offset >>= 1)
        {
            if (tid < offset)
            {
                __shared__s_x[tid] += __shared__s_x[tid + offset];
                __shared__s_y[tid] += __shared__s_y[tid + offset];
                __shared__s_x2[tid] += __shared__s_x2[tid + offset];
                __shared__s_xy[tid] += __shared__s_xy[tid + offset];
            }
            __syncthreads();
        }

        y = __shared__s_y[tid];
        x = __shared__s_x[tid];
        x2 = __shared__s_x2[tid];
        xy = __shared__s_xy[tid];
        for (int offset = 16; offset > 0; offset >>= 1)
        {
            y += __shfl_down_sync(0xffffffff,y, offset);
            x += __shfl_down_sync(0xffffffff,x, offset);
            x2 += __shfl_down_sync(0xffffffff,x2, offset);
            xy += __shfl_down_sync(0xffffffff,xy, offset);
        }

        if (tid == 0)
        {
            atomicAdd(&d_z[0], x);
            atomicAdd(&d_z[1], y);
            atomicAdd(&d_z[2], x2);
            atomicAdd(&d_z[3], xy);

        }

        if(n == 0) {
            double fenzi = d_z[3] - (d_z[0] * d_z[1] / N);
            double fenmu = d_z[2] - (d_z[0] * d_z[0] / N);
            d_z[4] = fenzi / fenmu;
            d_z[5] = (d_z[1] / N) - ((d_z[0] / N) * d_z[4]);

        }
}
int main(int argc, char* argv[]) {
        double *a = (double *) malloc(sizeof(double)*N);
        double *a2 = (double *) malloc(sizeof(double)*N);
        double *b = (double *) malloc(sizeof(double)*10);
        for (int x = 0 ;x < N;x ++) {
            a[x] = 14.21412 + 1.1245 * x;
            a2[x] = 124.21452 + 123.0523 * x;
        }
        double *d_x;
        double *d_y;
        hipMalloc((void **)&d_x,sizeof(double)*N);
        hipMalloc((void **)&d_y,sizeof(double)*N);
        hipMemcpy(d_x,a,sizeof(double)*N,hipMemcpyHostToDevice);
        hipMemcpy(d_y,a2,sizeof(double)*N,hipMemcpyHostToDevice);
        const int block_size = 128;
        const int grid_size =  (N + block_size - 1) / block_size;
        hipEvent_t start ;
        hipEvent_t stop ;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);

        sum<<<grid_size,block_size>>>(d_x,d_y);

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float elapsed_time = 0;
        hipEventElapsedTime(&elapsed_time,start,stop);
        printf("Time = %g ms .\n",elapsed_time);

        hipDeviceSynchronize();
        hipMemcpyFromSymbol(b,HIP_SYMBOL(d_z),sizeof(double)*10);
        printf("x sum:  %f \n",b[0]);
        printf("y sum: %f \n",b[1]);
        printf("x2 sum: %f \n",b[2]);
        printf("xy sum: %f \n",b[3]);
        printf("Result: y =  %f * x + (%f) \n",b[4],b[5]);
}
