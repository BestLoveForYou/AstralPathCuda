
#include <hip/hip_runtime.h>
#include<stdio.h>
using namespace std;
//现在开始全局变量书写
     const int N = 10;//global
//现在开始全局内存书写
__device__  int d_x[N] =  {};//length:N
__device__  int d[2] =  {};//length:2
__global__ void reduce() {
        const int tid = threadIdx.x;
        const int bid = blockIdx.x;
        const int n = bid * blockDim.x + tid;
__shared__ int __shared__s_y[128];
        if (n < N) {
            __shared__s_y[tid] = __ldg(&d_x[n]);
        }
        __syncthreads();

        for (int offset = blockDim.x >> 1; offset >= 32; offset >>= 1)
        {
            if (tid < offset)
            {
                __shared__s_y[tid] += __shared__s_y[tid + offset];
            }
            __syncthreads();
        }

        int y = __shared__s_y[tid];
        for (int offset = 16; offset > 0; offset >>= 1)
        {
            y += __shfl_down_sync(0xffffffff, y, offset);
        }

        if (tid == 0)
        {
            atomicAdd(&d[0], y);
        }
}
int main(int argc, char* argv[]) {
int ha[N] =  {};//length:N
        for(int x = 0; x < N ; ++x) {
            ha[x] = 1;
        }
        ha[0] = atoi(argv[1]);

        hipMemcpyToSymbol(HIP_SYMBOL(d_x),ha,sizeof(int)*N);

        const int block_size = 128;
        const int grid_size =  (N + block_size - 1) / block_size;

        reduce<<<grid_size,block_size>>>();


        int device_id = 0;

        hipSetDevice(device_id);
        hipDeviceProp_t prop ;
        hipGetDeviceProperties(&prop,device_id);
        printf("Device id:                                 %d\n",
                device_id);
        printf("Device name:                               %s\n",
                prop.name);
        printf("Compute capability:                        %d.%d\n",
                prop.major, prop.minor);
        printf("Amount of global memory:                   %g GB\n",
                prop.totalGlobalMem / (1024.0 * 1024 * 1024));
        printf("Amount of constant memory:                 %g KB\n",
                prop.totalConstMem  / 1024.0);
        printf("Maximum grid size:                         %d %d %d\n",
                prop.maxGridSize[0],
                prop.maxGridSize[1], prop.maxGridSize[2]);
        printf("Maximum block size:                        %d %d %d\n",
                prop.maxThreadsDim[0], prop.maxThreadsDim[1],
                prop.maxThreadsDim[2]);
        printf("Number of SMs:                             %d\n",
                prop.multiProcessorCount);
        printf("Maximum amount of shared memory per block: %g KB\n",
                prop.sharedMemPerBlock / 1024.0);
        printf("Maximum amount of shared memory per SM:    %g KB\n",
                prop.sharedMemPerMultiprocessor / 1024.0);
        printf("Maximum number of registers per block:     %d K\n",
                prop.regsPerBlock / 1024);
        printf("Maximum number of registers per SM:        %d K\n",
                prop.regsPerMultiprocessor / 1024);
        printf("Maximum number of threads per block:       %d\n",
                prop.maxThreadsPerBlock);
        printf("Maximum number of threads per SM:          %d\n",
                prop.maxThreadsPerMultiProcessor);

        hipMemcpyFromSymbol(ha,HIP_SYMBOL(d),sizeof(int)*2);
        printf("c=%d\n",ha[0]);
}
