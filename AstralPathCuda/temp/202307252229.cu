#include "hip/hip_runtime.h"
#include<stdio.h>
#include <hip/hip_runtime.h>
using namespace std;
//现在开始全局变量书写
     int N = 1;//global
//现在开始全局内存书写
__device__  int d_z[10] =  {};//length:10
__global__ void sum(int *d_x,int N) {
        const int tid = threadIdx.x;
        const int bid = blockIdx.x;
__shared__ int __shared__s_x[128];
        const int n = bid * blockDim.x + tid;
        int x = 0;
        if(n < N) {
            __shared__s_x[n] = d_x[n];
        }
        __syncthreads();
        for (int offset = blockDim.x >> 1; offset >= 32; offset >>= 1)
        {
            if (tid < offset)
            {
                __shared__s_x[tid] += __shared__s_x[tid + offset];
            }
            __syncthreads();
        }
        x = __shared__s_x[tid];
        for (int offset = 16; offset > 0; offset >>= 1)
        {
            x += __shfl_down_sync(0xffffffff,x, offset);
        }


        if (tid == 0)
        {

            atomicAdd(&d_z[0], x);
        }
    }
    @Override
    __device__ void main() {
        N = argc - 1;
        int *a = (int*) malloc(sizeof(int)*N);
        int *b = (int *) malloc(sizeof(int)*10);
        for (int x = 0 ;x < N;x ++) {
            a[x] = atoi(argv[x + 1]);
        }
        int *d_x;
        hipMalloc((void **)&d_x,sizeof(int)*N);
        hipMemcpy(d_x,a,sizeof(int)*N,hipMemcpyHostToDevice);
        const int block_size = 128;
        const int grid_size =  (N + block_size - 1) / block_size;
        hipEvent_t start ;
        hipEvent_t stop ;
        hipEventCreate(&start);
        hipEventCreate(&stop);
        hipEventRecord(start);

        sum<<<grid_size,block_size>>>(d_x,N);
        hipDeviceSynchronize();

        hipEventRecord(stop);
        hipEventSynchronize(stop);
        float elapsed_time = 0;
        hipEventElapsedTime(&elapsed_time,start,stop);
        printf("Time:%g ms\n",elapsed_time);

        hipDeviceSynchronize();
        hipMemcpyFromSymbol(b,HIP_SYMBOL(d_z),sizeof(int)*10);
        printf("x:%d \n",b[0]);

        free(a);
        free(b);
        hipFree(d_x);
}
int main(int argc, char* argv[]) {
}
