
#include <hip/hip_runtime.h>
#include<stdio.h>
using namespace std;
//现在开始全局变量书写
     const int N = 1000;//global
//现在开始全局内存书写
__global__ void test(int *h,int *b) {
        printf("(%d,%d)\n",threadIdx.x,threadIdx.y);
        b[threadIdx.x] = h[threadIdx.x];
    }
int main(int argc, char* argv[]) {
        hipStream_t stream_1 ;
        hipStream_t stream_2 ;

        hipStreamCreate(&stream_1);
        hipStreamCreate(&stream_2);

        int M = sizeof(int)*N;
        int *h_x = (int*) malloc(M);
        for (int x = 0;x < N;x ++) {
            h_x[x] = 1;
        }
        int *d_x1;
        int *d_y1;
        hipMalloc((void **)&d_x1,M);
        hipMemcpy(d_x1,h_x,M,hipMemcpyHostToDevice);
        hipMalloc((void **)&d_y1,M);

        const int block_size = 128;
        const int grid_size =  (N + block_size - 1) / block_size;

        test<<<grid_size,block_size,0,stream_1>>>(d_x1,d_y1);//tags:<<<grid_size,block_size,0,stream_1>>>
        test<<<grid_size,block_size,0,stream_2>>>(d_x1,d_y1);//tags:<<<grid_size,block_size,0,stream_2>>>

        hipStreamSynchronize(stream_1);
        hipStreamSynchronize(stream_2);

        hipStreamDestroy(stream_1);
        hipStreamDestroy(stream_2);
}
