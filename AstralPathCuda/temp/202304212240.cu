
#include <hip/hip_runtime.h>
#include<stdio.h>
using namespace std;
//现在开始全局变量书写
     const int N = 128;//global
//现在开始全局内存书写
__device__  int d[128] =  {};//length:128
__device__  int e[128] =  {};//length:128
__global__ void hello() {
        const int x = threadIdx.x;
        const int y = threadIdx.y;
        int n = blockDim.x * blockIdx.x + threadIdx.x;
        int a = d[n];
        int b = e[n];
        d[n] = a*b;
}
int main(int argc, char* argv[]) {
int ha[128] =  {};//length:128
        for(int x = 0; x < 128 ; ++x) {
            ha[x] = 5;
        }
        hipMemcpyToSymbol(HIP_SYMBOL(d),ha,sizeof(int)*2);
        for(int x = 0; x < 128 ; ++x) {
            ha[x] = 7;
        }
        hipMemcpyToSymbol(HIP_SYMBOL(e),ha,sizeof(int)*2);
        const int block_size = 128;
        const int grid_size = N / block_size;

        hello<<<grid_size,block_size>>>();
        hipMemcpyFromSymbol(ha,HIP_SYMBOL(d),sizeof(int)*2);
        printf("c=%d\n",ha[0]);
}
